#include "hip/hip_runtime.h"


// #include "cuda_common.cuh"
#include <stdio.h>

#include <stdlib.h>
#include <time.h>

#include <cstring>

__global__ void sum_array_gpu(int *a, int *b, int * c, int size){

    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < size){
        c[gid] = a[gid] + b[gid];
    }
}

void sum_array_cpu(int *a, int *b, int * c, int size){
    for (int i = 0; i < size; i++){
        c[i] = a[i] + b[i];
    }
}

void compare_arrays(int *a, int *b, int size){
    for (int i=0; i < size; i++){
        if (a[i] != b[i]){
            printf("Arrays are different \n");
            return;
        }
    }
    printf("Arrays are same \n");
}


int main(){

    int size = 1 << 25;

    int block_size = 1024;

    hipError_t error;

    int NO_BYTES = size * sizeof(int);

    int * h_a, * h_b, * gpu_results, * h_c;

    h_a = (int*)malloc(NO_BYTES);
    h_b = (int*)malloc(NO_BYTES);
    h_c = (int*)malloc(NO_BYTES);
    gpu_results = (int*)malloc(NO_BYTES);


    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; i++){
        h_a[i]= (int)(rand() & 0xFF);
    }

    for (int i = 0; i < size; i++){
        h_b[i]= (int)(rand() & 0xFF);
    }

    memset(gpu_results, 0, NO_BYTES);
    memset(h_c, 0, NO_BYTES);
    
    clock_t cpu_start, cpu_end;
    cpu_start = clock();

    sum_array_cpu(h_a, h_b, h_c, size);

    cpu_end = clock();
    int * d_a, * d_b, * d_c;

    error = hipMalloc((int **) &d_a, NO_BYTES);
    if (error != hipSuccess){
        fprintf(stderr, "Error : %s \n", hipGetErrorString(error));
    }
    hipMalloc((int **) &d_b, NO_BYTES);
    hipMalloc((int **) &d_c, NO_BYTES);

    clock_t htod_start, htod_end;

    htod_start = clock();

    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

    htod_end = clock();


    dim3 block(block_size);
    dim3 grid(size/block.x + 1);

    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum_array_gpu <<<grid, block>>> (d_a, d_b, d_c, size);
    hipDeviceSynchronize();

    gpu_end = clock();

    clock_t dtoh_start, dtoh_end;
    dtoh_start = clock();
    hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
    dtoh_end = clock();


    compare_arrays(gpu_results, h_c, size);
 
    printf("time taken in cpus: %4.6f \n" ,(double)((double)(cpu_end - cpu_start))/ CLOCKS_PER_SEC);
    printf("time taken in htod: %4.6f \n" ,(double)((double)(htod_end - htod_start))/ CLOCKS_PER_SEC);
    printf("time taken in dtoh: %4.6f \n" ,(double)((double)(dtoh_end - dtoh_start))/ CLOCKS_PER_SEC);
    printf("time taken in gpus: %4.6f \n" ,(double)((double)(gpu_end - gpu_start))/ CLOCKS_PER_SEC);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(gpu_results);
    
    hipDeviceReset();


    return 0;
}