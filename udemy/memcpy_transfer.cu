
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>


__global__ void mem_trs_test(int * input){
    int gid = blockIdx.x * blockDim.x + threadIdx.x ;
    printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
}

int main(){
    int size = 128;
    int byte_size = size * sizeof(int);

    int * h_input;
    h_input = (int*)malloc(byte_size);  // malloc returns void pointer so casting it to integer pointer


    time_t t;

    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++){
        h_input[i] = (int)(rand() & 0xff0);
    }

    int * d_input;

    hipMalloc((void**)&d_input, byte_size);

    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

    dim3 block(64);
    dim3 grid(2);

    mem_trs_test <<<grid, block>>> (d_input);
    hipDeviceSynchronize();
    hipFree(d_input);
    free(h_input);

    hipDeviceReset();
    return 0;
}