#include "hip/hip_runtime.h"



#include <stdio.h>


__global__ void hello_cuda(){
    printf("hello_world \n");
}


int main() {
    hello_cuda <<<1,20>>>();
    hipDeviceSynchronize();


    hipDeviceReset();
    return 0;
}
